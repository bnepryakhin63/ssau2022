﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <cmath>
#include <time.h>

using namespace std;

class Matrix
{
private:
	int razmer_matric;
	double** matrica;
public:
	Matrix(int razmer)
	{
		razmer_matric = razmer;
		matrica = new double* [razmer];
		for (int i = 0; i < razmer; i++) {
			matrica[i] = new double[razmer];
		}
	}
	void free_memory();
	void random_values();
	int get_size();
	void set_value(int line, int column, double value);
	double get_value(int line, int column);
	Matrix transposition();
	void write_matrix();
};

void Matrix::free_memory()
{
	delete[] matrica;
}

void Matrix::random_values()
{
	int razmer = razmer_matric;
	for (int i = 0; i < razmer; i++) {
		for (int j = 0; j < razmer; j++) {
			matrica[i][j] = 2 * ((double)rand() / (double)RAND_MAX) - 1;
		}
	}
}

int Matrix::get_size()
{
	return razmer_matric;
}

void Matrix::set_value(int line, int column, double value)
{
	matrica[line][column] = value;
}

double Matrix::get_value(int line, int column)
{
	return matrica[line][column];
}

Matrix Matrix::transposition()
{
	int razmer = razmer_matric;
	Matrix At(razmer);

	for (int i = 0; i < razmer; i++) {
		for (int j = 0; j < razmer; j++) {
			At.set_value(i, j, matrica[j][i]);
		}
	}
	return At;
}

void Matrix::write_matrix()
{
	int razmer = razmer_matric;
	for (int i = 0; i < razmer; i++) {
		cout << "| ";
		for (int j = 0; j < razmer; j++) {
			cout << matrica[i][j] << " ";
		}
		cout << "|" << endl;
	}
	cout << endl;
}

Matrix multiplication(Matrix A, Matrix B)
{
	int razmer = A.get_size();
	double sum;
	Matrix C(razmer);

	for (int k = 0; k < razmer; k++) {
		for (int l = 0; l < razmer; l++) {
			sum = 0;
			for (int m = 0; m < razmer; m++) {
				sum = sum + A.get_value(k, m) * B.get_value(m, l);
			}
			C.set_value(k, l, sum);
		}
	}

	return C;
}

void copy_values(Matrix matrica1, double matrica2[])
{
	int razmer = matrica1.get_size();
	for (int i = 0; i < razmer * razmer; i++) {
		matrica2[i] = matrica1.get_value(i / razmer, i % razmer);
	}
}

__global__ void deviceTransposition(double* matrica1, double* matrica2, int* razmer)
{
	unsigned long long int i = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned long long int siatka = blockDim.x * gridDim.x;
	for (i; i < razmer[0] * razmer[0]; i += siatka) {
		matrica2[i] = matrica1[(i % razmer[0]) * razmer[0] + i / razmer[0]];
	}
}

__global__ void deviceMultiplication(double* matrica1, double* matrica2, double* matrica3, int* razmer)
{
	unsigned long long int i = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned long long int j = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned long long int siatkax = blockDim.x * gridDim.x;
	unsigned long long int siatkay = blockDim.y * gridDim.y;
	double sum;
	for (i; i < razmer[0] * razmer[0]; i += siatkax) {
		sum = 0;
		for (j; j < razmer[0]; j += siatkay) {
			sum = sum + matrica1[((i / razmer[0]) * razmer[0]) + j] * matrica2[i % razmer[0] + j * razmer[0]];
		}
		matrica3[i] = sum;
	}
}

__global__ void deviceAddition(double* matrica1, double* matrica2, double* matrica3, int* razmer)
{
	unsigned long long int i = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned long long int siatka = blockDim.x * gridDim.x;
	for (i; i < razmer[0] * razmer[0]; i += siatka) {
		matrica3[i] = matrica2[i] + matrica1[i];
	}
}

int main()
{
	srand(time(NULL));
	while (true) {
		int razmer[1];
		cout << "Zadajte razmer matricy: ";
		cin >> razmer[0];

		Matrix A(razmer[0]);
		Matrix B(razmer[0]);
		Matrix C(razmer[0]);

		A.random_values();
		B.random_values();

		/*cout << "Matrica A:" << endl;
		A.write_matrix();
		cout << "Matrica B:" << endl;
		B.write_matrix();*/

		clock_t start;
		double duration_on_CPU;
		start = clock();

		C = multiplication(A, B);
		/*cout << "Matrica C na CPU:" << endl;
		C.write_matrix();*/

		duration_on_CPU = 1000 * (clock() - start) / CLOCKS_PER_SEC;

		cout << "Dlya umnozheniya matric A*B na CPU potrebovalos': " << duration_on_CPU << " ms." << endl;

		double duration_on_CPU1;
		start = clock();

		double* A1 = new double[razmer[0] * razmer[0]];
		double* B1 = new double[razmer[0] * razmer[0]];
		double* C1 = new double[razmer[0] * razmer[0]];

		copy_values(A, A1);
		copy_values(B, B1);

		double* dev_A;
		double* dev_At;
		double* dev_B;
		double* dev_Bt;
		double* dev_C;
		double* dev_Ct;
		double* dev_res;
		double* dev_res1;
		int* dev_razmer;

		int razmerBloka = 1024;
		int liczbaBlokow = (razmer[0] * razmer[0] + razmerBloka - 1) / razmerBloka;
		int sization = razmer[0] * razmer[0] * sizeof(double);

		hipMalloc((void**)&dev_razmer, sizeof(int));
		hipMalloc((void**)&dev_A, sization);
		hipMalloc((void**)&dev_At, sization);
		hipMalloc((void**)&dev_B, sization);
		hipMalloc((void**)&dev_Bt, sization);
		hipMalloc((void**)&dev_C, sization);
		hipMalloc((void**)&dev_Ct, sization);
		hipMalloc((void**)&dev_res, sization);
		hipMalloc((void**)&dev_res1, sization);

		hipMemcpy(dev_A, A1, sization, hipMemcpyHostToDevice);
		hipMemcpy(dev_B, B1, sization, hipMemcpyHostToDevice);
		hipMemcpy(dev_razmer, razmer, sizeof(int), hipMemcpyHostToDevice);
		clock_t start2;
		double duration_on_GPU;
		start2 = clock();

		deviceMultiplication<<<liczbaBlokow, razmerBloka>>>(dev_A, dev_B, dev_C, dev_razmer);
		hipDeviceSynchronize();

		duration_on_GPU = 1000 * (clock() - start2) / CLOCKS_PER_SEC;
		hipMemcpy(C1, dev_C, sization, hipMemcpyDeviceToHost);
		cout << "Dlya umnozheniya matric A*B na GPU potrebovalos': " << duration_on_GPU << " ms." << endl;

		/*cout << "matrica C na GPU:" << endl;
		for (int i = 0; i < razmer[0] * razmer[0]; i++) {
			if (i % razmer[0] == 0) cout << endl;
			cout << C1[i] << " | ";
		}
		cout << endl;*/

		double duration_on_GPU1;
		start2 = clock();

		deviceTransposition<<<liczbaBlokow, razmerBloka>>>(dev_A, dev_At, dev_razmer);
		hipDeviceSynchronize();
		deviceTransposition<<<liczbaBlokow, razmerBloka>>>(dev_B, dev_Bt, dev_razmer);
		hipDeviceSynchronize();
		deviceMultiplication<<<liczbaBlokow, razmerBloka>>>(dev_B, dev_Bt, dev_res, dev_razmer);
		hipDeviceSynchronize();
		deviceTransposition<<<liczbaBlokow, razmerBloka>>>(dev_C, dev_Ct, dev_razmer);
		hipDeviceSynchronize();
		deviceMultiplication<<<liczbaBlokow, razmerBloka>>>(dev_C, dev_Ct, dev_res1, dev_razmer);
		hipDeviceSynchronize();

	
		cout << "Poslednij element matricy C: " << C1[razmer[0] * razmer[0] - 1] << endl;
		cout << endl;
		double stosunek;
		if (duration_on_GPU != 0) {
			stosunek = duration_on_CPU / duration_on_GPU;
			cout << "Pervyj raschet na GPU v " << stosunek << " raza bystree." << endl;
		}
		else {
			cout << "Pervyj raz na GPU raven nulyu, nevozmozhno vychislit' sootnoshenie" << endl;
		}
		if (duration_on_GPU1 != 0) {
			stosunek = duration_on_CPU1 / duration_on_GPU1;
			cout << "Vtoroj raschet na GPU v " << stosunek << " raza bystree." << endl;
		}
		else {
			cout << "Vtoroj raz na GPU raven nulyu, nevozmozhno vychislit' sootnoshenie" << endl;
		}
		cout << endl;

		double max = 0;
		for (int i = 0; i < razmer[0] * razmer[0]; i++) {
			if (abs(C.get_value(i / razmer[0], i % razmer[0]) - C1[i]) > max) max = abs(C.get_value(i / razmer[0], i % razmer[0]) - C1[i]);
		}
		if (max == 0) cout << "Net nikakoj raznicy mezhdu znazheniymi C s CPU i GPU" << endl;
		else cout << "V meste naibol'shego raskhozhdeniya znazheniya C otlichayutsya na " << max << endl;


		A.free_memory();
		B.free_memory();
		C.free_memory();
		hipFree(dev_razmer);
		hipFree(dev_A);
		hipFree(dev_B);
		hipFree(dev_C);
		hipFree(dev_res);
		delete[] A1;
		delete[] B1;
		delete[] C1;
	}
	return 0;
}